#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const int *Params, const int *st, const int *id, const float *dataraw, float *WU, float const pm){
  int nt0, tidx, tidy, bid, ind, NT, Nchan,N_spike;
  float xsum = 0.0f; 
  NT            = (int) Params[0];
  N_spike       = (int) Params[2];
  nt0           = (int) Params[3];
  Nchan         = (int) Params[4];
  
  tidx 		= threadIdx.x;
  tidy 		= threadIdx.y;
  bid 		= blockIdx.x;
  
  for(ind=0; ind<N_spike;ind++)
      if (id[ind]==bid){
		  tidy 		= threadIdx.y;
		  while (tidy<Nchan){	
			xsum = dataraw[st[ind]+tidx + NT * tidy]; 
  			WU[tidx+tidy*nt0 + nt0*Nchan * bid] = pm*WU[tidx+tidy*nt0 + nt0*Nchan * bid] +  (1-pm)* xsum;
			tidy+=blockDim.y;
		  }
	  }
}
//////////////////////////////////////////////////////////////////////////////////////////


/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  int *Params, *d_Params;
  int blocksPerGrid, N_spike, nt0;
  float *dpm;
  float pm;
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params        = (int*) mxGetData(prhs[0]);
  dpm           = (float*)mxGetData(prhs[5]);
  blocksPerGrid	= (int) Params[1];
  N_spike       = (int) Params[2];
  nt0           = (int) Params[3];
  pm            = (float)dpm[0];
  hipMalloc(&d_Params,      sizeof(int)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(int)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);
  
  /* collect input GPU variables*/
  mxGPUArray const *dataraw;
  const float      *d_dataraw;
  dataraw       = mxGPUCreateFromMxArray(prhs[1]);
  d_dataraw     = (float const *)(mxGPUGetDataReadOnly(dataraw));

  float *d_dWU;
  mxGPUArray *dWU;
  dWU       = mxGPUCopyFromMxArray(prhs[4]);
  d_dWU     = (float *)(mxGPUGetData(dWU));
  
  
  /* allocate new GPU variables*/
  int *d_st,*d_id;
  int *t,*id;
  t = (int*)mxGetData(prhs[2]);
  id = (int*)mxGetData(prhs[3]);

  hipMalloc(&d_st,    N_spike * sizeof(int));
  hipMalloc(&d_id,    N_spike * sizeof(int));

  
  hipMemcpy(d_st,t,N_spike *   sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_id,id,N_spike *   sizeof(int),hipMemcpyHostToDevice);

  
  dim3 block(nt0, 1024/nt0);
  average_snips<<<blocksPerGrid,block>>>(d_Params, d_st, d_id, d_dataraw, d_dWU,pm);
 
  plhs[0] 	= mxGPUCreateMxArrayOnGPU(dWU);
  int   *x,*idt;
  const mwSize dimst[] 	= {N_spike ,1}; 
  plhs[1] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  plhs[2] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  x = (int*) mxGetData(plhs[1]);
  hipMemcpy(x,   d_st, N_spike * sizeof(int), hipMemcpyDeviceToHost);
  idt = (int*) mxGetData(plhs[2]);
  hipMemcpy(idt,   d_id, N_spike * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dpm);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_Params);
  mxGPUDestroyGPUArray(dataraw);
  mxGPUDestroyGPUArray(dWU); 
  
}