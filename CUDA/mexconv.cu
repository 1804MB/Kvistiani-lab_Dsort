#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int Nthreads = 1024;
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const int *Params, const float *data, const float *W, float *conv_sig){    
  __shared__ float  sW[81], sdata[(Nthreads+81)]; 
  float x,a,b;
  int tid, nt0, tid0, bid, i, nid, NT, Nfilt;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  Nfilt    	=   (int) Params[1];
  NT      	=   (int) Params[0];
  nt0       =   (int) Params[2];
  
  if(tid<nt0)
      sW[tid]= W[tid%nt0 + (bid + Nfilt * (tid/nt0))* nt0];
  __syncthreads();
  
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){
	  if (tid<nt0) sdata[tid%nt0 + (tid/nt0)*(Nthreads+nt0)] = 
			data[tid0 + tid%nt0+ NT*(bid + Nfilt*(tid/nt0))];
     
          sdata[tid + nt0+nid*(Nthreads+nt0)] = data[nt0+tid0 + tid+ NT*(bid +nid*Nfilt)];	  
	  __syncthreads();
      
	  
          x = 0.0f;
          a = 0.0f;
          b = 0.0f;
		  #pragma unroll 4
          for(i=0;i<nt0;i++){
              a    += sW[i + nid*nt0]*sW[i + nid*nt0];
              b    += sdata[i+tid + nid*(Nthreads+nt0)]*sdata[i+tid + nid*(Nthreads+nt0)];
              x    += sW[i + nid*nt0] * sdata[i+tid + nid*(Nthreads+nt0)];
           }
             x = x/(sqrt(a*b));
	  
      conv_sig[tid0  + tid + NT*bid]   = x;
      
      tid0+=Nthreads;
      __syncthreads();
  }
}
/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  int *Params, *d_Params;
  int blocksPerGrid, NT;
  int const threadsPerBlock = 1024;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();
 mxGPUArray *B;
  /* read Params and copy to GPU */
  Params        = (int*) mxGetData(prhs[0]);
  NT            = (int) Params[0];
  blocksPerGrid	= (int) Params[1];
  hipMalloc(&d_Params,      sizeof(int)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(int)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);
  /* collect input GPU variables*/
  mxGPUArray const  *W, *data; 
  const float      *d_W,*d_data;
  float *d_dout;
  const mwSize dimst1[] 	= {NT,blocksPerGrid}; 
  B = mxGPUCreateGPUArray(2,dimst1,mxSINGLE_CLASS, mxREAL,MX_GPU_DO_NOT_INITIALIZE);
  d_dout = (float *)(mxGPUGetData(B));

  W             = mxGPUCreateFromMxArray(prhs[2]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  data        	= mxGPUCreateFromMxArray(prhs[1]);
  d_data        = (float const *)(mxGPUGetDataReadOnly(data));
  
    
   Conv1D<<<blocksPerGrid,threadsPerBlock>>>(d_Params, d_data, d_W, d_dout);  
   plhs[0] = mxGPUCreateMxArrayOnGPU(B);

  
  hipFree(d_Params); 
  mxGPUDestroyGPUArray(B);
  mxGPUDestroyGPUArray(data);
  mxGPUDestroyGPUArray(W); 
 
  
}